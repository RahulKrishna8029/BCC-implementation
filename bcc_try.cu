#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
using namespace std;

__global__ void sub_bfs(int *vertices, int *edges, int *frontier, int *distances, int *visited, int *search, int *safe , int ui, int *next_frontier, int n) {
    
  int tidx = blockIdx.x * blockDim.x + threadIdx.x;
  tidx = tidx + 1;

  if(tidx>n){
    return;
  }

  if(frontier[tidx]) {
      next_frontier[tidx] = 0;
      visited[tidx] = 1;

      for(int i=vertices[tidx] ; i<vertices[tidx+1] && *safe==0; i++) {
          if(visited[edges[i]] == 0) {
              next_frontier[edges[i]] = true; 
              *search = 1;
              if(distances[edges[i]] < distances[ui]) {*safe = 1; break;}
          }
      }
  }
  // if(hipSuccess != hipDeviceSynchronize()) {
  //   return;
  // }
  
}

__global__ void cut_vertex(int *vertices, int *edges, int n, int *distances, int root, int *cut_vertices, int *unsafe_vertices) {
  int tidx = blockIdx.x * blockDim.x + threadIdx.x;
  tidx = tidx + 1;

  if(tidx >n) 
  return;

  int *isSafe;
  hipMalloc((void **)&isSafe, sizeof(int));
  int temp = 0;
  *isSafe = 0;

  int *visited;
    hipMalloc((void **)&visited , (sizeof(int) * (n+2)));

    // make a frontier array
    int *frontier;
    hipMalloc((void **)&frontier , (sizeof(int) * (n+2)));

    int *next_frontier;
  hipMalloc((void **)&next_frontier , (sizeof(int) * (n+2)));


  for(int i=vertices[tidx] ; i<vertices[tidx+1] ; i++) {
      if(distances[edges[i]] <= distances[tidx]) continue;
      int v = tidx;
      int src = edges[i];

      //truncated_bfs<<<1,1>>>(edges[i], tidx, vertices, edges, n, distances, isSafe);
      {
            for(int i=1 ; i<=n ; i++) {
                visited[i] = 0;
                frontier[i] = 0;
                next_frontier[i] = 0;
            }

        // mark v as visited
            visited[v] = 1;
        // push src into the frontier
            frontier[src] = 1;

            int threadsPerBlock = 1024;
            int blocksPerGrid = (n + 1023)/1024;

            int temp1 = 0;
            int temp2 = 0;
            int *search, *safe;

            hipMalloc((void **)&safe, sizeof(int));
            hipMalloc((void **)&search, sizeof(int));
            *search = 1;
            *safe = 0;

            while ( (*search)==1 && (*safe) == 0) 
            { 
                *search = 0;
                sub_bfs<<<blocksPerGrid, threadsPerBlock>>>(vertices, edges, frontier, distances, visited, search, safe, src, next_frontier, n);
                hipDeviceSynchronize();
                for(int i=0 ; i<=n ; i++) {
                  frontier[i] = next_frontier[i];
                  next_frontier[i] = 0;
              }
                if(tidx == root) *safe = 0;
            }

            *isSafe = *safe;
      }
      hipDeviceSynchronize();
      // if safe, continue
      // else make the status as true and break;
      if(*isSafe == 0 && tidx != root) {
        cut_vertices[tidx] = 1;
        unsafe_vertices[src] = tidx;
      }

      if(tidx == root) {
        for(int i=vertices[root] ; i<vertices[root+1] ; i++) {
            if(visited[edges[i]] == 0) {
                cut_vertices[root] = 1;
                unsafe_vertices[edges[i]] = root;
            }
        }
      }
  }

  hipDeviceSynchronize();
}

__global__ void bfs(int *vertices, int *edges, int n, int *distances, int *level, int *flag) {
  int tidx = blockIdx.x * blockDim.x + threadIdx.x;
  tidx = tidx + 1;

  if(tidx > n || distances[tidx] != (*level)) return;

  for(int i=vertices[tidx] ; i<vertices[tidx+1] ; i++) {
    if(distances[edges[i]] == 10000000) {
      distances[edges[i]] = *level + 1;
      *flag = 1;
    }
  }
}

__device__ int dfs(int u, int v, int flag, int num, int*vertices, int*component, int*edges, int*edge_visit, int com_idx) {
  int min_vertex = u;

  for(int i=vertices[u] ; i<vertices[u+1] ; i++) {
    if(edge_visit[i] == 0) {
      if(flag == 1) {
        component[i] = num;
      }
      edge_visit[i] = 1;
      if(edges[i] != v)
      min_vertex = min(min_vertex, dfs(edges[i], v, flag, num, vertices, component, edges,edge_visit, i));
    }
  }

  return min_vertex;
}

__global__ void bcc(int *vertices, int *edges, int *distances, int *edge_visit, int*edge_visit2, int *component, int level, int *unsafe_vertices, int n) {
  int tidx = blockIdx.x * blockDim.x + threadIdx.x;
  tidx = tidx + 1;

  if(tidx > n || distances[tidx] != level || unsafe_vertices[tidx] == -1) 
  return;

  // get the minimum vertex number present in a bcc component
  int flag = 0;
  int min_vertex = dfs(tidx, unsafe_vertices[tidx], flag, 0, vertices, component, edges, edge_visit, -1);
  printf("tidx: %d min: %d  \n", tidx, min_vertex);
  // hipDeviceSynchronize();
  // assign that minimum vertex to each node of that component
  flag = 1;
  dfs(tidx, unsafe_vertices[tidx], flag, min_vertex, vertices, component, edges, edge_visit2, -1);
 // hipDeviceSynchronize();

}

int main() {
  int threadsPerBlock = 1024, blocksPerGrid;
  int n;
  cout << "enter the number of vertices\n";
  scanf("%d", &n);

  int src;
  cout << "enter the source\n";
  cin >> src;

  blocksPerGrid = (n + 1023)/1024;

  vector<vector<int>> edgeList(n);

  printf("enter the neighbours of 1 (end with a -1) followed by the neighbours of 2 (end with a -1) and so on till n\n");

  int cnt=0, edgeCnt=0;

  while(cnt < n) {
    int temp;
    cin >> temp;

    if(temp == -1) {
      cnt++;
      continue;
    }
    
    edgeCnt++;

    edgeList[cnt].push_back(temp);
  }

  int *vertices, *dvertices;
  vertices = (int *) malloc(sizeof(int) * (n+2));
  hipMalloc((void **)&dvertices, sizeof(int) * (n+2));

  int *edges, *dedges;
  edges = (int *) malloc(sizeof(int) * (edgeCnt + 2));
  hipMalloc((void **)&dedges, sizeof(int) * (edgeCnt + 2));

  int k=1;
  for(int i=0 ; i<=n ; i++) {
    vertices[i+1] = k;
    if(i == n) break;

    for(int j=0 ; j<edgeList[i].size() ; j++) {
      edges[k++] = edgeList[i][j];
    }
  }


  int *distances, *ddistances;

  distances = (int *) malloc(sizeof(int) * (n+2));
  hipMalloc((void **)&ddistances, sizeof(int) * (n+2));

  for(int i=1 ; i<=n ; i++) {
    distances[i] = 10000000;
  }

  distances[src] = 0;

  int *flag, *level;
  int *dflag, *dlevel;

  flag = (int *) (sizeof(int));
  level = (int *) (sizeof(int));

  hipMalloc((void **)&dflag, sizeof(int));
  hipMalloc((void **)&dlevel, sizeof(int));

  hipMemcpy(ddistances, distances, (n+2) * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dvertices, vertices, (n+2) * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dedges, edges, (edgeCnt + 2) * sizeof(int), hipMemcpyHostToDevice);

  int temp_level = 0, temp_flag = 1;

  level = &temp_level;
  flag = &temp_flag;

  cout << endl;

  while(*flag) {

    *flag = 0;
    hipMemcpy(dflag, flag, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dlevel, level, sizeof(int), hipMemcpyHostToDevice);

    bfs<<<blocksPerGrid, threadsPerBlock>>>(dvertices, dedges, n , ddistances, dlevel, dflag);
    hipDeviceSynchronize();
    *level = *level + 1;
    hipMemcpy(flag, dflag, sizeof(int), hipMemcpyDeviceToHost);

  }

  hipMemcpy(distances, ddistances, sizeof(int) * (n+2), hipMemcpyDeviceToHost);
  
  int max_level = 0;
  cout << "\ndistances array after parallel bfs\n";
  for(int i=1 ; i<=n ; i++) {
    cout << distances[i] << " ";
    max_level = max(max_level, distances[i]);
  }
  cout << endl;

  int *cut_vertices , *dcut_vertices;
  cut_vertices = (int *) malloc((n+2)*sizeof(int));
  hipMalloc((void**)&dcut_vertices , (n+2)*sizeof(int));

  for(int i = 1 ; i <= n ; i++)
    cut_vertices[i] = 0;

  int *unsafe_vertices, *dunsafe_vertices;
  unsafe_vertices = (int *) malloc((n+2) * sizeof(int));
  hipMalloc((void **)&dunsafe_vertices, (n+2)*sizeof(int));

  for(int i=0 ; i<=n ; i++) {
    unsafe_vertices[i] = -1;
  }
  
  hipMemcpy(dunsafe_vertices , unsafe_vertices , (n+2)*sizeof(int) , hipMemcpyHostToDevice);
  hipMemcpy(dcut_vertices , cut_vertices , (n+2)*sizeof(int) , hipMemcpyHostToDevice);
  cut_vertex<<<blocksPerGrid, threadsPerBlock>>>(dvertices, dedges, n, ddistances, src, dcut_vertices, dunsafe_vertices);
  hipDeviceSynchronize();
  hipMemcpy(cut_vertices , dcut_vertices , (n+2)*sizeof(int) , hipMemcpyDeviceToHost);
  hipMemcpy(unsafe_vertices , dunsafe_vertices , (n+2)*sizeof(int) , hipMemcpyDeviceToHost);


  cout << "For each vertex i from 1 to n, prints 1 if its a cutvertex else 0\n";
  for(int i=1 ; i<=n ; i++) {
    cout << cut_vertices[i] << " ";
  }
  cout << endl;

  for(int i=1 ; i<=n ; i++) {
    cout << unsafe_vertices[i] << " ";
  }
  cout << endl;

  int *edge_visit, *dedge_visit, *dedge_visit2;
  edge_visit = (int *) malloc((edgeCnt + 2) * sizeof(int));
  hipMalloc((void**)&dedge_visit , (edgeCnt + 2) * sizeof(int));
  hipMalloc((void**)&dedge_visit2 , (edgeCnt + 2) * sizeof(int));

  for(int i=0 ; i<=edgeCnt+1 ; i++) {
    edge_visit[i] = 0;
  }

  int *component, *dcomponent;
  component = (int *) malloc((edgeCnt+2) * sizeof(int));
  hipMalloc((void **)&dcomponent, (edgeCnt+2) * sizeof(int));
  for(int i=0 ; i<=edgeCnt ; i++) {
    component[i] = 0;
  }

  hipMemcpy(dcomponent , component, (edgeCnt+2) * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dedge_visit , edge_visit, (edgeCnt+2) * sizeof(int) , hipMemcpyHostToDevice);
  hipMemcpy(dedge_visit2 , edge_visit, (edgeCnt+2) * sizeof(int) , hipMemcpyHostToDevice);
  while(max_level >= 0) {

    bcc<<<blocksPerGrid, threadsPerBlock>>>(dvertices, dedges, ddistances, dedge_visit, dedge_visit2, dcomponent, max_level, dunsafe_vertices, n);
    hipDeviceSynchronize();
    max_level--;

  }
  hipMemcpy(component , dcomponent, (edgeCnt+2) * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(edge_visit , dedge_visit, (edgeCnt+2) * sizeof(int) , hipMemcpyDeviceToHost);

  cout << endl;
  // for(int i=1 ; i<=n ; i++) {
  //   cout << component[i] << " ";
  // }

  cout << "For each edge (u,v) from 1 to n, prints the component number of the edge\n";
  for(int i=1 ; i<=n ; i++) {
    for(int j=vertices[i] ; j<vertices[i+1] ; j++) {
       // cout << i << " " << edges[j] << " " << component[j] << endl;
       if(distances[i] > distances[edges[j]]) {
         cout << i << " " << edges[j] << " " << component[j] << endl;
       }
       else if(distances[i] == distances[edges[j]] && i < edges[j]) {
         cout << i << " " << edges[j] << " " << component[j] << endl;
       }
    }
  }

  cout << endl;
  return 0;
}


// ex: 2 3 -1 1 4 5 -1 1 6 7 -1 2 5 8 -1 2 4 9 -1 3 -1 3 -1 4 -1 5 10 -1 9 -1
// 2 3 -1 1 3 6 -1 1 2 6 4 5 -1 3 12 13 -1 3 6 -1 2 3 5 7 8 -1 6 8 -1 6 7 9 11 -1 8 11 -1 11 -1 8 9 10 -1 4 13 14 -1 4 12 14 -1 12 13 15 -1 13 14 16 17 -1 15 17 -1 15 16 -1