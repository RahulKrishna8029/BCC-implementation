#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
using namespace std;


__global__ void sub_bfs(int *vertices, int *edges, int *frontier, int *distances, int *visited, int *search, int *safe , int ui, int *next_frontier, int n) {
    
  int tidx = blockIdx.x * blockDim.x + threadIdx.x;
  tidx = tidx + 1;

  if(tidx>n){
    return;
  }

  if(frontier[tidx]) {
      next_frontier[tidx] = 0;
      visited[tidx] = 1;

      for(int i=vertices[tidx] ; i<vertices[tidx+1] && *safe==0; i++) {
          if(visited[edges[i]] == 0) {
              next_frontier[edges[i]] = true; 
              *search = 1;
              if(distances[edges[i]] < distances[ui]) {*safe = 1; break;}
          }
      }
  }
  // if(hipSuccess != hipDeviceSynchronize()) {
  //   return;
  // }
  
}

__global__ void truncated_bfs(int src, int v, int *vertices, int *edges, int n, int *distances, int *isSafe) {
   // make a visited array
    int *visited;
    visited = (int *) (sizeof(int) * (n+2));

    // make a frontier array
    int *frontier;
    frontier = (int *) (sizeof(int) * (n+2));

    for(int i=1 ; i<=n ; i++) {
        visited[i] = 0;
        frontier[i] = 0;
    }

   // mark v as visited
    visited[v] = 1;
   // push src into the frontier
    frontier[src] = 1;

    int threadsPerBlock = 1024;
    int blocksPerGrid = (n + 1023)/1024;

    int temp1 = 0;
    int temp2 = 0;
    int *search, *safe;

    hipMalloc((void **)&safe, sizeof(int));
    hipMalloc((void **)&search, sizeof(int));
    *search = 1;
    *safe = 0;

    while ( (*search)==1 && (*safe) == 0) 
    { 
        *search = 0;
          sub_bfs<<<blocksPerGrid, threadsPerBlock>>>(vertices, edges, frontier, distances, visited, search, safe, src, frontier, n);
          if(hipSuccess != hipDeviceSynchronize()) {
            return;
          }
    }
    // if(hipSuccess != hipDeviceSynchronize()) {
    //   return;
    // }
    *isSafe = *safe;
}

__global__ void cut_vertex(int *vertices, int *edges, int n, int *distances, int root, int *cut_vertices) {
  int tidx = blockIdx.x * blockDim.x + threadIdx.x;
  tidx = tidx + 1;

  if(tidx >n) 
  return;

  int *isSafe;
  hipMalloc((void **)&isSafe, sizeof(int));
  int temp = 0;
  *isSafe = 0;

  int *visited;
  hipMalloc((void **)&visited , (sizeof(int) * (n+2)));

  // make a frontier array
  int *frontier;
  hipMalloc((void **)&frontier , (sizeof(int) * (n+2)));

  int *next_frontier;
  hipMalloc((void **)&next_frontier , (sizeof(int) * (n+2)));

  for(int i=vertices[tidx] ; i<vertices[tidx+1] ; i++) {
      if(distances[edges[i]] <= distances[tidx]) continue;
      int v = tidx;
      int src = edges[i];

      //truncated_bfs<<<1,1>>>(edges[i], tidx, vertices, edges, n, distances, isSafe);

      for(int i=1 ; i<=n ; i++) {
          visited[i] = 0;
          frontier[i] = 0;
          next_frontier[i] = 0;
      }

  // mark v as visited
      visited[v] = 1;
  // push src into the frontier
      frontier[src] = 1;

      int threadsPerBlock = 1024;
      int blocksPerGrid = (n + 1023)/1024;

      int temp1 = 0;
      int temp2 = 0;
      int *search, *safe;

      hipMalloc((void **)&safe, sizeof(int));
      hipMalloc((void **)&search, sizeof(int));
      *search = 1;
      *safe = 0;

      while ( (*search)==1 && (*safe) == 0) 
      { 
          *search = 0;
          sub_bfs<<<blocksPerGrid, threadsPerBlock>>>(vertices, edges, frontier, distances, visited, search, safe, src, next_frontier, n);
          if(hipSuccess != hipDeviceSynchronize()) {
            return;
          }
          for(int i=0 ; i<=n ; i++) {
              frontier[i] = next_frontier[i];
              next_frontier[i] = 0;
          }
          if(tidx == root) *safe = 0;
          // if(hipSuccess != hipDeviceSynchronize()) {
          //   return;
          // }
      }

      *isSafe = *safe;

      if(hipSuccess != hipDeviceSynchronize()) {
        return;
      }
      // if safe, continue
      // else make the status as true and break;
      if(*isSafe == 0 && tidx != root) {
        cut_vertices[tidx] = 1;
        break;
      }
  }

  if(tidx==root) {
    for(int i=vertices[root] ; i<vertices[root+1] ; i++) {
        if(visited[edges[i]] == 0) {
            cut_vertices[root] = 1;
            break;
        }
    }
  }
  // if(hipSuccess != hipDeviceSynchronize()) {
  //   return;
  // }
}

__global__ void bfs(int *vertices, int *edges, int n, int *distances, int *level, int *flag) {
  int tidx = blockIdx.x * blockDim.x + threadIdx.x;
  tidx = tidx + 1;

  if(tidx > n || distances[tidx] != (*level)) return;

  for(int i=vertices[tidx] ; i<vertices[tidx+1] ; i++) {
    if(distances[edges[i]] == 10000000) {
      distances[edges[i]] = *level + 1;
      *flag = 1;
    }
  }
}

int main() {
  int threadsPerBlock = 1024, blocksPerGrid;
  int n;
  cout << "enter the number of vertices\n";
  scanf("%d", &n);

  int src;
  cout << "enter the source\n";
  cin >> src;

  blocksPerGrid = (n + 1023)/1024;

  vector<vector<int>> edgeList(n);

  printf("enter the neighbours of 1 (end with a -1) followed by the neighbours of 2 (end with a -1) and so on till n\n");

  int cnt=0, edgeCnt=0;

  while(cnt < n) {
    int temp;
    cin >> temp;

    if(temp == -1) {
      cnt++;
      continue;
    }
    
    edgeCnt++;

    edgeList[cnt].push_back(temp);
  }
  cout<<"a\n";
  int *vertices, *dvertices;
  vertices = (int *) malloc(sizeof(int) * (n+2));
  hipMalloc((void **)&dvertices, sizeof(int) * (n+2));

  int *edges, *dedges;
  edges = (int *) malloc(sizeof(int) * (edgeCnt + 2));
  hipMalloc((void **)&dedges, sizeof(int) * (edgeCnt + 2));

  int k=1;
  for(int i=0 ; i<=n ; i++) {
    vertices[i+1] = k;
    if(i == n) break;

    for(int j=0 ; j<edgeList[i].size() ; j++) {
      edges[k++] = edgeList[i][j];
    }
  }


  int *distances, *ddistances;

  distances = (int *) malloc(sizeof(int) * (n+2));
  hipMalloc((void **)&ddistances, sizeof(int) * (n+2));

  for(int i=1 ; i<=n ; i++) {
    distances[i] = 10000000;
  }

  distances[src] = 0;

  int *flag, *level;
  int *dflag, *dlevel;

  flag = (int *) (sizeof(int));
  level = (int *) (sizeof(int));

  hipMalloc((void **)&dflag, sizeof(int));
  hipMalloc((void **)&dlevel, sizeof(int));

  hipMemcpy(ddistances, distances, (n+2) * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dvertices, vertices, (n+2) * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dedges, edges, (edgeCnt + 2) * sizeof(int), hipMemcpyHostToDevice);

  int temp_level = 0, temp_flag = 1;

  level = &temp_level;
  flag = &temp_flag;

  cout << endl;

  while(*flag) {

    *flag = 0;
    hipMemcpy(dflag, flag, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dlevel, level, sizeof(int), hipMemcpyHostToDevice);

    bfs<<<blocksPerGrid, threadsPerBlock>>>(dvertices, dedges, n , ddistances, dlevel, dflag);
    if(hipSuccess != hipDeviceSynchronize()) {
      return;
    }
    *level = *level + 1;
    hipMemcpy(flag, dflag, sizeof(int), hipMemcpyDeviceToHost);

  }

  // if(hipSuccess != hipDeviceSynchronize()) {
  //   return;
  // }
  hipMemcpy(distances, ddistances, sizeof(int) * (n+2), hipMemcpyDeviceToHost);
  
  cout << "\ndistances array after parallel bfs\n";
  for(int i=1 ; i<=n ; i++) {
    cout << distances[i] << " ";
  }
  cout << endl;

  int *cut_vertices , *dcut_vertices;
  cut_vertices = (int *) malloc((n+2)*sizeof(int));
  hipMalloc((void**)&dcut_vertices , (n+2)*sizeof(int));

  for(int i = 1 ; i <= n ; i++)
    cut_vertices[i] = 0;
  
  hipMemcpy(dcut_vertices , cut_vertices , (n+2)*sizeof(int) , hipMemcpyHostToDevice);
  cut_vertex<<<blocksPerGrid, threadsPerBlock>>>(dvertices, dedges, n, ddistances, src, dcut_vertices);
  // if(hipSuccess != hipDeviceSynchronize()) {
  //   return;
  // }
  hipDeviceSynchronize();
  hipMemcpy(cut_vertices , dcut_vertices , (n+2)*sizeof(int) , hipMemcpyDeviceToHost);


  cout << "For each vertex i from 1 to n, prints 1 if its a cutvertex else 0\n";
  for(int i=1 ; i<=n ; i++) {
    cout << cut_vertices[i] << " ";
  }
  cout << endl;

  cout << endl;
  return 0;
}

// ex: 2 3 -1 1 4 5 -1 1 6 7 -1 2 5 8 -1 2 4 9 -1 3 -1 3 -1 4 -1 5 10 -1 9 -1
// 2 3 -1 1 3 6 -1 1 2 6 4 5 -1 3 12 13 -1 3 6 -1 2 3 5 7 8 -1 6 8 -1 6 7 9 11 -1 8 11 -1 11 -1 8 9 10 -1 4 13 14 -1 4 12 14 -1 12 13 15 -1 13 14 16 17 -1 15 17 -1 15 16 -1